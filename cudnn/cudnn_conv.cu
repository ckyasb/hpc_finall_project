#include "winograd.cuh"
#include <hipDNN.h>
#include <iostream>
#include <vector>

// Macro to check cuDNN API call status
#define CUDNN_CHECK(status) { \
    if (status != HIPDNN_STATUS_SUCCESS) { \
        std::cerr << "cuDNN error in " << __FILE__ << ":" << __LINE__ \
                  << " : " << hipdnnGetErrorString(status) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

void cudnn_winograd_conv(thrust::device_vector<float>& image,
                         thrust::device_vector<float>& filter, 
                         thrust::device_vector<float>& out,
                         int H, int W, int C, int K, int N) {

    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    // 1. Define Tensor Descriptors
    hipdnnTensorDescriptor_t input_desc, output_desc;
    hipdnnFilterDescriptor_t filter_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filter_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, K, C, 3, 3));
    
    const int outH = H - 2;
    const int outW = W - 2;
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, K, outH, outW));

    // 2. Define Convolution Descriptor
    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv_desc, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // 3. Directly force the Winograd algorithm, bypassing search
    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;
    
    // 4. Get workspace size for the forced algorithm
    size_t workspace_bytes = 0;
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(handle, input_desc, filter_desc, conv_desc, output_desc, algo, &workspace_bytes));

    void* d_workspace = nullptr;
    if (workspace_bytes > 0) {
        hipMalloc(&d_workspace, workspace_bytes);
    }

    // 5. Attempt to execute the convolution
    const float alpha = 1.0f;
    const float beta = 0.0f;
    
    hipdnnStatus_t status = hipdnnConvolutionForward(handle,
                                      &alpha,
                                      input_desc, image.data().get(),
                                      filter_desc, filter.data().get(),
                                      conv_desc,
                                      algo,
                                      d_workspace, workspace_bytes,
                                      &beta,
                                      output_desc, out.data().get());
    
    // If the forced algorithm fails, print an error and zero out the result
    if (status != HIPDNN_STATUS_SUCCESS) {
        std::cerr << "cuDNN error: Forced WINOGRAD algorithm failed for this layer with status: " 
                  << hipdnnGetErrorString(status) << std::endl;
        hipMemset(out.data().get(), 0, out.size() * sizeof(float));
    }

    // 6. Clean up resources
    if (d_workspace) {
        hipFree(d_workspace);
    }
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(handle);
}

